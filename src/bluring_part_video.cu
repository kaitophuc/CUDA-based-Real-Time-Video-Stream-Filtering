#include "hip/hip_runtime.h"
#include "../lib/bluring_part_video.hpp"
#include <iostream>
#include <string>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>

__managed__ int *blur_x;
__managed__ int *blur_y;
__managed__ int *distance;
bool enable = true;

__host__ void CheckCudaError(const std::string& error_message) {
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "Error: " << hipGetErrorString(err) << std::endl;
    std::cerr << error_message << std::endl;
    exit(1);
  }
}

/*__global__ void Convert(uchar* dr_in, uchar* dg_in, uchar* db_in, uchar* dr_out, uchar* dg_out, uchar* db_out, 
                        int idx, int width, int height, int x, int y) {
    int col = blockIdx.x * TILE_DIM + threadIdx.x;
    int row = blockIdx.y * TILE_DIM + threadIdx.y;
    int num_pixels = width * height;
    // Shared memory for the input image. Use for tiling the image to avoid bank conflicts.
    __shared__ uchar dr_in_shared[TILE_DIM][TILE_DIM];
    __shared__ uchar dg_in_shared[TILE_DIM][TILE_DIM];
    __shared__ uchar db_in_shared[TILE_DIM][TILE_DIM];
    if (col < width && row < height) {
      dr_in_shared[threadIdx.y][threadIdx.x] = dr_in[idx * num_pixels + row * width + col];
      dg_in_shared[threadIdx.y][threadIdx.x] = dg_in[idx * num_pixels + row * width + col];
      db_in_shared[threadIdx.y][threadIdx.x] = db_in[idx * num_pixels + row * width + col];
    } else {
      dr_in_shared[threadIdx.y][threadIdx.x] = 0;
      dg_in_shared[threadIdx.y][threadIdx.x] = 0;
      db_in_shared[threadIdx.y][threadIdx.x] = 0;
    }

    __syncthreads();

    if (col < width && row < height) {
      if ((col - x) * (col - x) + (row - y) * (row - y) <= DISTANCE * DISTANCE) {
        double pix_val_r = 0.00;
        double pix_val_g = 0.00;
        double pix_val_b = 0.00;
        for (int fRow = -FILTER_RADIUS; fRow <= FILTER_RADIUS; fRow++) {
            for (int fCol = -FILTER_RADIUS; fCol <= FILTER_RADIUS; fCol++) {
                int tileRow = threadIdx.y + fRow;
                int tileCol = threadIdx.x + fCol;
                if (tileRow >= 0 && tileRow < TILE_DIM && tileCol >= 0 && tileCol < TILE_DIM) {
                    pix_val_r += dr_in_shared[tileRow][tileCol] * Gaussian[fRow + FILTER_RADIUS][fCol + FILTER_RADIUS];
                    pix_val_g += dg_in_shared[tileRow][tileCol] * Gaussian[fRow + FILTER_RADIUS][fCol + FILTER_RADIUS];
                    pix_val_b += db_in_shared[tileRow][tileCol] * Gaussian[fRow + FILTER_RADIUS][fCol + FILTER_RADIUS];
                }
                else {
                    int imageRow = row + fRow;
                    int imageCol = col + fCol;
                    if (imageRow >= 0 && imageRow < height && imageCol >= 0 && imageCol < width) {
                        pix_val_r += dr_in[idx * num_pixels + imageRow * width + imageCol] * Gaussian[fRow + FILTER_RADIUS][fCol + FILTER_RADIUS];
                        pix_val_g += dg_in[idx * num_pixels + imageRow * width + imageCol] * Gaussian[fRow + FILTER_RADIUS][fCol + FILTER_RADIUS];
                        pix_val_b += db_in[idx * num_pixels + imageRow * width + imageCol] * Gaussian[fRow + FILTER_RADIUS][fCol + FILTER_RADIUS];
                    }
                }
            }
        }
        dr_out[idx * num_pixels + row * width + col] = static_cast<uchar>(pix_val_r);// - dr_in[idx * num_pixels + row * width + col]));
        dg_out[idx * num_pixels + row * width + col] = static_cast<uchar>(pix_val_g);// - dg_in[idx * num_pixels + row * width + col]));
        db_out[idx * num_pixels + row * width + col] = static_cast<uchar>(pix_val_b);// - db_in[idx * num_pixels + row * width + col]));
      } else {
        dr_out[idx * num_pixels + row * width + col] = dr_in[idx * num_pixels + row * width + col];
        dg_out[idx * num_pixels + row * width + col] = dg_in[idx * num_pixels + row * width + col];
        db_out[idx * num_pixels + row * width + col] = db_in[idx * num_pixels + row * width + col];

      }
    }
}*/

__global__ void Convert(uchar* dr_in, uchar* dg_in, uchar* db_in, uchar* dr_out, uchar* dg_out, uchar* db_out, 
                        int idx, int width, int height) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int num_pixels = width * height;
  // Shared memory for the input image. Use for tiling the image to avoid bank conflicts.
  __shared__ uchar dr_in_shared[TILE_DIM][TILE_DIM];
  __shared__ uchar dg_in_shared[TILE_DIM][TILE_DIM];
  __shared__ uchar db_in_shared[TILE_DIM][TILE_DIM];
  
  if (col < width && row < height) {
    dr_in_shared[threadIdx.y][threadIdx.x] = dr_in[idx * num_pixels + row * width + col];
    dg_in_shared[threadIdx.y][threadIdx.x] = dg_in[idx * num_pixels + row * width + col];
    db_in_shared[threadIdx.y][threadIdx.x] = db_in[idx * num_pixels + row * width + col];
  } else {
    dr_in_shared[threadIdx.y][threadIdx.x] = 0;
    dg_in_shared[threadIdx.y][threadIdx.x] = 0;
    db_in_shared[threadIdx.y][threadIdx.x] = 0;
  }

  __syncthreads();

  if (col < width && row < height) {
    if ((col - *blur_x) * (col - *blur_x) + (row - *blur_y) * (row - *blur_y) <= (*distance) * (*distance)) {
      int pix_val_r = 0;
      int pix_val_g = 0;
      int pix_val_b = 0;
      int pixels = 0;

      // Get the average of the surrounding pixels
      for (int f_row = -BLUR_SIZE; f_row <= BLUR_SIZE; f_row++) {
        for (int f_col = -BLUR_SIZE; f_col <= BLUR_SIZE; f_col++) {
          int tile_row = threadIdx.y + f_row;
          int tile_col = threadIdx.x + f_col;
          if (tile_row >= 0 && tile_row < TILE_DIM && tile_col >= 0 && tile_col < TILE_DIM) {
            pix_val_r += dr_in_shared[tile_row][tile_col];
            pix_val_g += dg_in_shared[tile_row][tile_col];
            pix_val_b += db_in_shared[tile_row][tile_col];
            ++pixels;
          } else {
            int i = row + f_row;
            int j = col + f_col;
            if (i >= 0 && i < height && j >= 0 && j < width) {
              pix_val_r += dr_in[idx * num_pixels + i * width + j];
              pix_val_g += dg_in[idx * num_pixels + i * width + j];
              pix_val_b += db_in[idx * num_pixels + i * width + j];
              ++pixels;
            }
          }
        }
      }

      dr_out[idx * num_pixels + row * width + col] = static_cast<uchar>(pix_val_r / pixels);
      dg_out[idx * num_pixels + row * width + col] = static_cast<uchar>(pix_val_g / pixels);
      db_out[idx * num_pixels + row * width + col] = static_cast<uchar>(pix_val_b / pixels);
    } else {
      dr_out[idx * num_pixels + row * width + col] = dr_in[idx * num_pixels + row * width + col];
      dg_out[idx * num_pixels + row * width + col] = dg_in[idx * num_pixels + row * width + col];
      db_out[idx * num_pixels + row * width + col] = db_in[idx * num_pixels + row * width + col];
    }
  }
}

__host__ void AllocateHostMemory(uchar** hr_in, uchar** hg_in, uchar** hb_in, uchar** hr_out, uchar** hg_out, 
                                 uchar** hb_out, int num_pixels) {
  *hr_in = static_cast<uchar*>(malloc(num_pixels * NUM_FRAMES * sizeof(uchar)));
  *hg_in = static_cast<uchar*>(malloc(num_pixels * NUM_FRAMES * sizeof(uchar)));
  *hb_in = static_cast<uchar*>(malloc(num_pixels * NUM_FRAMES * sizeof(uchar)));
  *hr_out = static_cast<uchar*>(malloc(num_pixels * NUM_FRAMES * sizeof(uchar)));
  *hg_out = static_cast<uchar*>(malloc(num_pixels * NUM_FRAMES * sizeof(uchar)));
  *hb_out = static_cast<uchar*>(malloc(num_pixels * NUM_FRAMES * sizeof(uchar)));
}

__host__ void AllocateDeviceMemory(uchar** dr_in, uchar** dg_in, uchar** db_in, uchar** dr_out, uchar** dg_out, 
                                   uchar** db_out, int num_pixels) {
  hipMalloc(dr_in, num_pixels * NUM_FRAMES * sizeof(uchar));
  CheckCudaError("Error allocating device memory for dr_in");
  hipMalloc(dg_in, num_pixels * NUM_FRAMES * sizeof(uchar));
  CheckCudaError("Error allocating device memory for dg_in");
  hipMalloc(db_in, num_pixels * NUM_FRAMES * sizeof(uchar));
  CheckCudaError("Error allocating device memory for db_in");
  hipMalloc(dr_out, num_pixels * NUM_FRAMES * sizeof(uchar));
  CheckCudaError("Error allocating device memory for dr_out");
  hipMalloc(dg_out, num_pixels * NUM_FRAMES * sizeof(uchar));
  CheckCudaError("Error allocating device memory for dg_out");
  hipMalloc(db_out, num_pixels * NUM_FRAMES * sizeof(uchar));
  CheckCudaError("Error allocating device memory for db_out");
}

// Copy the input image from the host to the device
__host__ void CopyFromHostToDevice(uchar* hr_in, uchar* hg_in, uchar* hb_in, uchar* dr_in, uchar* dg_in, uchar* db_in, 
                                   int count, int width, int height) {
  int num_pixels = count * width * height;
  size_t size = num_pixels * sizeof(uchar);
  hipMemcpy(dr_in, hr_in, size, hipMemcpyHostToDevice);
  CheckCudaError("Error copying from host to device");
  hipMemcpy(dg_in, hg_in, size, hipMemcpyHostToDevice);
  CheckCudaError("Error copying from host to device");
  hipMemcpy(db_in, hb_in, size, hipMemcpyHostToDevice);
  CheckCudaError("Error copying from host to device");
}

// Copy the result from the device to the host
__host__ void CopyFromDeviceToHost(uchar* dr_out, uchar* dg_out, uchar* db_out, uchar* hr_out, uchar* hg_out, 
                                   uchar* hb_out, int count, int width, int height) {
  int num_pixels = count * width * height;
  size_t size = num_pixels * sizeof(uchar);
  hipMemcpy(hr_out, dr_out, size, hipMemcpyDeviceToHost);
  CheckCudaError("Error copying from device to host");
  hipMemcpy(hg_out, dg_out, size, hipMemcpyDeviceToHost);
  CheckCudaError("Error copying from device to host");
  hipMemcpy(hb_out, db_out, size, hipMemcpyDeviceToHost);
  CheckCudaError("Error copying from device to host");
}

// Free the device memory
__host__ void FreeDeviceMemory(uchar* dr_in, uchar* dg_in, uchar* db_in, uchar* dr_out, uchar* dg_out, uchar* db_out) {
  hipFree(dr_in);
  CheckCudaError("Error freeing device memory for dr_in");
  hipFree(dg_in);
  CheckCudaError("Error freeing device memory for dg_in");
  hipFree(db_in);
  CheckCudaError("Error freeing device memory for db_in");
  hipFree(dr_out);
  CheckCudaError("Error freeing device memory for dr_out");
  hipFree(dg_out);
  CheckCudaError("Error freeing device memory for dg_out");
  hipFree(db_out);
  CheckCudaError("Error freeing device memory for db_out");
}

__host__ void CleanUp() {
  hipDeviceReset();
  CheckCudaError("Error resetting device");
}

// Kernel to blur the image. Using dynamic parallelism to blur the image.
__global__ void Blur(uchar* dr_in, uchar* dg_in, uchar* db_in, uchar* dr_out, uchar* dg_out, uchar* db_out, 
                     int width, int height) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  dim3 block_size(TILE_DIM, TILE_DIM);
  dim3 grid_size((width + block_size.x - 1) / block_size.x, (height + block_size.y - 1) / block_size.y);
  Convert<<<grid_size, block_size>>>(dr_in, dg_in, db_in, dr_out, dg_out, db_out, idx, width, height);
}

__host__ void Execute(uchar* dr_in, uchar* dg_in, uchar* db_in, uchar* dr_out, uchar* dg_out, uchar* db_out, 
                      int count, int width, int height, int *x, int *y) {
  Blur<<<1, count>>>(dr_in, dg_in, db_in, dr_out, dg_out, db_out, width, height);
  CheckCudaError("Error executing kernel");
  hipDeviceSynchronize();
}

// Read the image from the file and store it in the host memory
__host__ void ReadImageFromFile(cv::Mat* image, uchar* hr_total, uchar* hg_total, uchar* hb_total, int count, 
                                int width, int height) {
  int num_pixels = width * height;

  #pragma omp parallel for collapse(2)
  for (int i = 0; i < height; i++) {
    for (int j = 0; j < width; j++) {
      cv::Vec3b pixel = image->at<cv::Vec3b>(i, j);
      hr_total[count * num_pixels + i * width + j] = pixel[2];
      hg_total[count * num_pixels + i * width + j] = pixel[1];
      hb_total[count * num_pixels + i * width + j] = pixel[0];
    }
  }
}

// Mouse callback function to get the coordinates of the mouse click
void OnMouse(int event, int x, int y, int, void* userdata) {
  cv::Mat* image = reinterpret_cast<cv::Mat*>(userdata);
  if (event == cv::EVENT_LBUTTONDOWN) {
    *blur_x = x;
    *blur_y = y;
    enable = true;
  } else if (event == cv::EVENT_RBUTTONDOWN) {
    *blur_x = -1;
    *blur_y = -1;
    enable = false;
  }
}

void DoJob(cv::Mat frame, int width, int height, int frames, int num_pixels, uchar* hr_in, uchar* hg_in, uchar* hb_in, 
           uchar* hr_out, uchar* hg_out, uchar* hb_out, uchar* dr_in, uchar* dg_in, uchar* db_in, 
           uchar* dr_out, uchar* dg_out, uchar* db_out, int count) {
  CopyFromHostToDevice(hr_in, hg_in, hb_in, dr_in, dg_in, db_in, count, width, height);
  CheckCudaError("Error copying from host to device");
  Execute(dr_in, dg_in, db_in, dr_out, dg_out, db_out, count, width, height, blur_x, blur_y);
  CheckCudaError("Error executing kernel");
  CopyFromDeviceToHost(dr_out, dg_out, db_out, hr_out, hg_out, hb_out, count, width, height);
  CheckCudaError("Error copying from device to host");

  cv::Mat output_image = cv::Mat::zeros(height, width, CV_8UC3);
  for (int idx = 0; idx < count; idx++) {
    #pragma omp parallel for collapse(2)
    for (int i = 0; i < height; i++) {
      for (int j = 0; j < width; j++) {
        cv::Vec3b pixel;
        pixel[2] = hr_out[idx * num_pixels + i * width + j];
        pixel[1] = hg_out[idx * num_pixels + i * width + j];
        pixel[0] = hb_out[idx * num_pixels + i * width + j];
        //output_image.at<cv::Vec3b>(i, j) = pixel;
        frame.at<cv::Vec3b>(i, j) = pixel;
      }
    }
    //cv::imshow("Blurred Image", frame);
  }
}
  

int main(int argc, char** argv) {
  // Allocate Unified Memory
  hipMallocManaged(&blur_x, sizeof(int));
  hipMallocManaged(&blur_y, sizeof(int));
  hipMallocManaged(&distance, sizeof(int));

  // Initialize variables
  *blur_x = -1;
  *blur_y = -1;
  *distance = 100;

  // Read the video file path from the command line
  if (argc < 2) {
    std::cerr << "Usage: " << argv[0] << " <video_file_path>" << std::endl;
    return -1;
  }

  std::string video_file_path = argv[1];

  // Load face detection model
  std::string modelConfiguration = "./models/deploy.prototxt";
  std::string modelWeights = "./models/res10_300x300_ssd_iter_140000.caffemodel";
  cv::dnn::Net net = cv::dnn::readNetFromCaffe(modelConfiguration, modelWeights);

  try {
    // Create a window to display the blurred image
    cv::VideoCapture cap;
    if (video_file_path == "0") {
      cap.open(0);
    } else {
      cap.open(video_file_path);
    }
    if (!cap.isOpened()) {
      std::cerr << "Error: Unable to open video file\n";
      return -1;
    }

    int width = cap.get(cv::CAP_PROP_FRAME_WIDTH);
    int height = cap.get(cv::CAP_PROP_FRAME_HEIGHT);
    int frames = cap.get(cv::CAP_PROP_FPS);
    int num_pixels = width * height;

    std::cout << "Width: " << width << " Height: " << height << " Frames: " << frames << std::endl;

    // Allocate device memory
    uchar *dr_in, *dg_in, *db_in, *dr_out, *dg_out, *db_out;
    AllocateDeviceMemory(&dr_in, &dg_in, &db_in, &dr_out, &dg_out, &db_out, num_pixels);

    // Allocate host memory
    uchar* hr_in; uchar* hg_in; uchar* hb_in; uchar* hr_out; uchar* hg_out; uchar* hb_out;
    AllocateHostMemory(&hr_in, &hg_in, &hb_in, &hr_out, &hg_out, &hb_out, num_pixels);

    cv::Mat frame;

    auto start = std::chrono::high_resolution_clock::now();

    while (true) {
      int count = 0;
      bool flag = true;
      // Read the video frames
      for (int i = 0; i < NUM_FRAMES; i++) {
        if (!cap.read(frame)) {
          flag = false;
          std::cerr << "Error: Unable to read video frame\n";
          auto end = std::chrono::high_resolution_clock::now();
          auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
          std::cout << "Time: " << duration.count() << " ms" << std::endl;
        }
        ReadImageFromFile(&frame, hr_in, hg_in, hb_in, count, width, height);
        ++count;
      }

      if (!enable) {
        *blur_x = -1;
        *blur_y = -1;
      }

      else {

        // Convert frame to blob
        cv::Mat blob = cv::dnn::blobFromImage(frame, 1.0, cv::Size(300, 300), cv::Scalar(104.0, 177.0, 123.0));

        // Set the input to the network
        net.setInput(blob);

        // Perform forward pass to get the face detections
        cv::Mat detections = net.forward();

        // Get the dimensions of the detections matrix
        const int numDetections = detections.size[2];
        const int numCoords = detections.size[3];

        // Get a pointer to the data in the detections matrix
        float* data = (float*)detections.ptr<float>(0);

        bool face_detected = false;
        // Loop over the detections
        for (int i = 0; i < numDetections; ++i) {
            float confidence = data[i * numCoords + 2];

            // If confidence is above a threshold, draw a rectangle around the face
            if (confidence > 0.5) {
                std::cout << confidence << std::endl;
                face_detected = true;
                int x1 = static_cast<int>(data[i * numCoords + 3] * frame.cols);
                int y1 = static_cast<int>(data[i * numCoords + 4] * frame.rows);
                int x2 = static_cast<int>(data[i * numCoords + 5] * frame.cols);
                int y2 = static_cast<int>(data[i * numCoords + 6] * frame.rows);

                // Ensure the rectangle coordinates are within the image boundaries
                x1 = std::max(0, std::min(x1, frame.cols - 1));
                y1 = std::max(0, std::min(y1, frame.rows - 1));
                x2 = std::max(0, std::min(x2, frame.cols - 1));
                y2 = std::max(0, std::min(y2, frame.rows - 1));

                // Apply blur to the detected face region
                if (x2 > x1 && y2 > y1) {
                    *blur_x = (x1 + x2) / 2;
                    *blur_y = (y1 + y2) / 2;
                    *distance = sqrt ((x2 - *blur_x) * (x2 - *blur_x) + (y2 - *blur_y) * (y2 - *blur_y));
                    DoJob(frame, width, height, frames, num_pixels, hr_in, hg_in, hb_in, hr_out, hg_out, hb_out, dr_in, dg_in, db_in, dr_out, dg_out, db_out, count);
      
                      // Draw a rectangle around the detected face
                    cv::rectangle(frame, cv::Point(x1, y1), cv::Point(x2, y2), cv::Scalar(0, 255, 0), 2); // Green rectangle with thickness 2
                    cv::imshow("Blurred Image", frame);
              
                }
            }
        }

        if (face_detected == 0) {
          *blur_x = -1;
          *blur_y = -1;
        }

      }


      // If no mouse click, display the original image
      if (*blur_x == -1 && *blur_y == -1) {
        cv::Mat output_image = cv::Mat::zeros(height, width, CV_8UC3);
        for (int idx = 0; idx < count; idx++) {
          #pragma omp parallel for collapse(2)
          for (int i = 0; i < height; i++) {
            for (int j = 0; j < width; j++) {
              cv::Vec3b pixel;
              pixel[2] = hr_in[idx * num_pixels + i * width + j];
              pixel[1] = hg_in[idx * num_pixels + i * width + j];
              pixel[0] = hb_in[idx * num_pixels + i * width + j];
              output_image.at<cv::Vec3b>(i, j) = pixel;
            }
          }
          cv::imshow("Blurred Image", output_image);
          if (cv::waitKey(1000 / frames) == 27) {
            flag = false;
            break;
          }
        }
      } 
      
      else {
        DoJob(frame, width, height, frames, num_pixels, hr_in, hg_in, hb_in, hr_out, hg_out, hb_out, dr_in, dg_in, db_in, dr_out, dg_out, db_out, count);
      }

      cv::setMouseCallback("Blurred Image", OnMouse, &frame);
      //std::cout << blur_x << " " << blur_y << std::endl;
      if (cv::waitKey(1000 / frames) == 27) break;
    }

    // measure execution time
    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start);
    std::cout << "Time: " << duration.count() << " ms" << std::endl;
  } 
  
  catch (const std::exception& e) {
    std::cerr << "Error: " << e.what() << std::endl;
    return 1;
  }

  return 0;
}
