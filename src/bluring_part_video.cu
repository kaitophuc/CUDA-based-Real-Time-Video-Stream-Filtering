// Main application file - orchestrates all blur kernels
#include "blur_common.hpp"
#include "blur_naive.hpp"
#include "blur_multistream.hpp"
#include "blur_cub.hpp"
#include "blur_prefix_sum.hpp"
#include <iomanip>
#include <thread>

int main(int argc, char** argv) {
  // Allocate Unified Memory for multiple faces
  hipMallocManaged(&blur_x, MAX_FACES * sizeof(int));
  hipMallocManaged(&blur_y, MAX_FACES * sizeof(int));
  hipMallocManaged(&distance, MAX_FACES * sizeof(int));
  hipMallocManaged(&num_faces, sizeof(int));

  // Initialize variables
  *num_faces = 0;
  for (int i = 0; i < MAX_FACES; i++) {
    blur_x[i] = -1;
    blur_y[i] = -1;
    distance[i] = DISTANCE;
  }

  // Read the video file path from the command line
  if (argc < 2) {
    std::cerr << "Usage: " << argv[0] << " <video_file_path> [mode]" << std::endl;
    std::cerr << "Modes: test (default), interactive, benchmark, webcam_benchmark" << std::endl;
    std::cerr << "  test: Test all kernels with video file" << std::endl;
    std::cerr << "  interactive: Choose a kernel for interactive mode" << std::endl;
    std::cerr << "  benchmark: Test all kernels then run best in interactive mode" << std::endl;
    std::cerr << "  webcam_benchmark: Test all kernels with webcam for 10 seconds each" << std::endl;
    return -1;
  }

  std::string video_file_path = argv[1];
  std::string mode = (argc >= 3) ? argv[2] : "interactive";

  // Initialize components
  cv::dnn::Net net = initializeFaceDetection();
  cv::VideoCapture cap = initializeVideoCapture(video_file_path);

  int width = cap.get(cv::CAP_PROP_FRAME_WIDTH);
  int height = cap.get(cv::CAP_PROP_FRAME_HEIGHT);
  int frames = cap.get(cv::CAP_PROP_FPS);
  int num_pixels = width * height;

  std::cout << "Resolution: " << width << "x" << height << " @ " << frames << " FPS" << std::endl;

  // Allocate memory
  uchar *h_buf, *d_buf;
  uchar *dr_in, *dg_in, *db_in, *dr_out, *dg_out, *db_out;
  AllocateDeviceMemory(&d_buf, &dr_in, &dg_in, &db_in, &dr_out, &dg_out, &db_out, num_pixels);

  uchar* hr_in; uchar* hg_in; uchar* hb_in; uchar* hr_out; uchar* hg_out; uchar* hb_out;
  AllocateHostMemory(&h_buf, &hr_in, &hg_in, &hb_in, &hr_out, &hg_out, &hb_out, num_pixels);

  // Initialize available kernels - now using modular implementations
  std::vector<KernelPerformance> kernels = {
    KernelPerformance("Naive CUDA", Blur_Naive),
    KernelPerformance("Multi-Stream CUDA", Blur_MultiStream),
    KernelPerformance("CUB Optimized", Blur_CUB),
    KernelPerformance("Brent-Kung Prefix Sum", Blur_Brent_Kung)
  };

  try {
    if (mode == "test" || mode == "benchmark") {
      // Test all kernels
      std::cout << "\n=== KERNEL PERFORMANCE COMPARISON ===" << std::endl;
      
      for (auto& kernel : kernels) {
        testKernel(kernel, cap, net, width, height, frames, num_pixels,
                  hr_in, hg_in, hb_in, hr_out, hg_out, hb_out,
                  dr_in, dg_in, db_in, dr_out, dg_out, db_out);
      }
      
      // Print summary
      std::cout << "\n=== PERFORMANCE SUMMARY ===" << std::endl;
      std::cout << std::setw(20) << "Kernel Name" << std::setw(15) << "Avg FPS" << std::setw(15) << "Realtime FPS" << std::setw(15) << "Total Time" << std::endl;
      std::cout << std::string(65, '-') << std::endl;
      
      for (const auto& kernel : kernels) {
        std::cout << std::setw(20) << kernel.name 
                  << std::setw(15) << std::fixed << std::setprecision(2) << kernel.avg_fps
                  << std::setw(15) << std::fixed << std::setprecision(2) << kernel.smoothed_fps
                  << std::setw(15) << std::fixed << std::setprecision(2) << kernel.total_time << "s" << std::endl;
      }
      
      // Find best performing kernel based on realtime smoothed FPS (what user actually sees)
      auto best_kernel = std::max_element(kernels.begin(), kernels.end(),
        [](const KernelPerformance& a, const KernelPerformance& b) {
          return a.smoothed_fps < b.smoothed_fps;
        });
      
      std::cout << "\nBest performing kernel: " << best_kernel->name 
                << " (Realtime: " << std::fixed << std::setprecision(2) << best_kernel->smoothed_fps 
                << " FPS, Average: " << best_kernel->avg_fps << " FPS)" << std::endl;
                
      if (mode == "benchmark") {
        runInteractiveMode(*best_kernel, cap, net, width, height, frames, num_pixels,
                          hr_in, hg_in, hb_in, hr_out, hg_out, hb_out,
                          dr_in, dg_in, db_in, dr_out, dg_out, db_out);
      }
      
    } else if (mode == "webcam_benchmark") {
      // Force webcam for benchmark
      if (video_file_path != "0") {
        std::cout << "Webcam benchmark mode: switching to webcam (0)" << std::endl;
        cap.release();
        cap = initializeVideoCapture("0");
        width = cap.get(cv::CAP_PROP_FRAME_WIDTH);
        height = cap.get(cv::CAP_PROP_FRAME_HEIGHT);
        frames = cap.get(cv::CAP_PROP_FPS);
        num_pixels = width * height;
        std::cout << "Webcam Resolution: " << width << "x" << height << " @ " << frames << " FPS" << std::endl;
      }
      
      // Benchmark all kernels with webcam
      std::cout << "\n=== WEBCAM KERNEL BENCHMARK (10 seconds each) ===" << std::endl;
      
      // Reset kernel performance data
      for (auto& kernel : kernels) {
        kernel.avg_fps = 0.0;
        kernel.total_time = 0.0;
        kernel.frame_count = 0;
      }
      
      for (auto& kernel : kernels) {
        benchmarkKernel(kernel, cap, net, width, height, frames, num_pixels,
                       hr_in, hg_in, hb_in, hr_out, hg_out, hb_out,
                       dr_in, dg_in, db_in, dr_out, dg_out, db_out,
                       10.0); // Benchmark for 10 seconds each
        
        // Small break between tests
        std::cout << "Preparing for next test in 2 seconds..." << std::endl;
        std::this_thread::sleep_for(std::chrono::seconds(2));
      }
      
      // Print summary
      std::cout << "\n=== WEBCAM BENCHMARK SUMMARY ===" << std::endl;
      std::cout << std::setw(20) << "Kernel Name" << std::setw(15) << "Total FPS" << std::setw(15) << "Total Frames" << std::setw(15) << "Test Duration" << std::endl;
      std::cout << std::string(65, '-') << std::endl;
      
      for (const auto& kernel : kernels) {
        std::cout << std::setw(20) << kernel.name 
                  << std::setw(15) << std::fixed << std::setprecision(2) << kernel.avg_fps
                  << std::setw(15) << kernel.frame_count
                  << std::setw(15) << std::fixed << std::setprecision(1) << kernel.total_time << "s" << std::endl;
      }
      
      // Find best performing kernel based on total FPS (for webcam, kernel-only timing is not separately measured)
      auto best_kernel = std::max_element(kernels.begin(), kernels.end(),
        [](const KernelPerformance& a, const KernelPerformance& b) {
          return a.avg_fps < b.avg_fps;
        });
      
      std::cout << "\nBest performing kernel: " << best_kernel->name 
                << " (" << std::fixed << std::setprecision(2) << best_kernel->avg_fps << " FPS)" << std::endl;
      
    } else if (mode == "interactive") {
      // Let user choose kernel
      std::cout << "\nAvailable kernels:" << std::endl;
      for (size_t i = 0; i < kernels.size(); i++) {
        std::cout << i + 1 << ". " << kernels[i].name << std::endl;
      }
      
      std::cout << "Choose kernel (1-" << kernels.size() << "): ";
      int choice;
      std::cin >> choice;
      
      if (choice < 1 || choice > static_cast<int>(kernels.size())) {
        std::cout << "Invalid choice, using first kernel" << std::endl;
        choice = 1;
      }
      
      runInteractiveMode(kernels[choice - 1], cap, net, width, height, frames, num_pixels,
                        hr_in, hg_in, hb_in, hr_out, hg_out, hb_out,
                        dr_in, dg_in, db_in, dr_out, dg_out, db_out);
    }

    // Cleanup
    std::cout << "\nCleaning up resources..." << std::endl;
    
    // Close OpenCV windows first
    cv::destroyAllWindows();
    
    // Release video capture before DNN cleanup
    if (cap.isOpened()) {
      cap.release();
    }
    
    // Clear DNN network to prevent memory warnings
    net = cv::dnn::Net();
    
    // Small delay to ensure OpenCV cleanup completes
    std::this_thread::sleep_for(std::chrono::milliseconds(100));
    
    // Free unified memory first (these may be used by kernels)
    if (blur_x) {
      hipFree(blur_x);
      blur_x = nullptr;
    }
    if (blur_y) {
      hipFree(blur_y);
      blur_y = nullptr;
    }
    if (distance) {
      hipFree(distance);
      distance = nullptr;
    }
    if (num_faces) {
      hipFree(num_faces);
      num_faces = nullptr;
    }
    
    // Free CUDA device memory
    if (d_buf) {
      hipFree(d_buf);
      d_buf = nullptr;
    }
    
    // Free CUDA host memory
    if (h_buf) {
      hipHostFree(h_buf);
      h_buf = nullptr;
    }
    
    // Synchronize before device reset
    hipDeviceSynchronize();
    
    // Reset CUDA device
    hipError_t resetError = hipDeviceReset();
    if (resetError != hipSuccess) {
      std::cerr << "Warning: CUDA device reset failed: " << hipGetErrorString(resetError) << std::endl;
    }
    
    std::cout << "Cleanup completed." << std::endl;
    
  } catch (const std::exception& e) {
    std::cerr << "Error: " << e.what() << std::endl;
    return 1;
  }

  return 0;
}
