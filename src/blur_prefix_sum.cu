#include "hip/hip_runtime.h"
#include "blur_cub.hpp"

__device__ void Brent_Kung_Scan(int* data, int n) {
    const int tid = threadIdx.x;

    for (int stride = 1; stride < n; stride <<= 1) {
        int index = (tid + 1) * stride * 2 - 1;
        if (index < n) {
            data[index] += data[index - stride];
        }
        __syncthreads();
    }


    for (int stride = n >> 1; stride > 0; stride >>= 1) {
        int index = (tid + 1) * stride * 2 - 1;
        if (index + stride < n) {
            int temp = data[index];
            data[index] = data[index + stride];
            data[index + stride] += temp;
        }
        __syncthreads();
    }
}

//==========================================================================//
template<int R, int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void BoxBlurHorizontal(
    const uchar* d_in,
    int width, int height, int pitch,
    int *hsum)
{
  constexpr int TILE = BLOCK_THREADS * ITEMS_PER_THREAD;
  int row = blockIdx.y;
  if (row >= height) return;
  int x0 = blockIdx.x * TILE;
  int valid = max(0, min(TILE, width - x0));

  // Shared memory for the entire block
  __shared__ int shared_data[TILE];
  
  // Step 1: Load data from all threads into shared memory
  #pragma unroll
  for (int i = 0; i < ITEMS_PER_THREAD; i++) {
    int local_idx = threadIdx.x * ITEMS_PER_THREAD + i;
    int global_idx = x0 + local_idx;
    
    if (local_idx < valid && global_idx < width) {
      shared_data[local_idx] = d_in[row * pitch + global_idx];
    } else {
      shared_data[local_idx] = 0;
    }
  }
  __syncthreads();
  
  // Step 2: Call Brent-Kung on the entire block's data
  Brent_Kung_Scan(shared_data, valid);
  
  // Step 3: Use prefix sums for box blur calculations
  #pragma unroll
  for (int i = 0; i < ITEMS_PER_THREAD; i++) {
    int local_idx = threadIdx.x * ITEMS_PER_THREAD + i;
    int global_idx = x0 + local_idx;
    
    if (local_idx < valid) {
      int left = max(0, global_idx - R);
      int right = min(width - 1, global_idx + R);
      int L = max(0, left - x0);
      int R_ = min(valid - 1, right - x0);
      
      int sum = 0;
      if (L <= R_) {
        // O(1) range sum using prefix array
        sum = shared_data[R_] - (L > 0 ? shared_data[L-1] : 0);
      }
      
      // Add boundary pixels outside the tile (rare case)
      for (int c = left; c < x0; c++) {
        sum += d_in[row * pitch + c];
      }
      for (int c = x0 + valid; c <= right; c++) {
        if (c < width) {
          sum += d_in[row * pitch + c];
        }
      }
      
      hsum[row * pitch + global_idx] = sum;
    }
  }
}

template<int R, int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void BoxBlurVertical(
    const int* hsum,
    const uchar* d_in,
    uchar* d_out,
    int width, int height, int pitch)
{
  constexpr int TILE = BLOCK_THREADS * ITEMS_PER_THREAD;

  int col = blockIdx.x;
  int y0 = blockIdx.y * TILE;
  int valid = max(0, min(TILE, height - y0));
  if (col >= width) return;

  __shared__ int shared_data[TILE];

  #pragma unroll
  for (int i = 0; i < ITEMS_PER_THREAD; i++) {
    int y_local = threadIdx.x * ITEMS_PER_THREAD + i;
    int y_global = y0 + y_local;
    if (y_local < valid && y_global < height) {
      shared_data[i] = hsum[y_global * pitch + col];
    }
    else {
      shared_data[i] = 0;
    }
  }
  __syncthreads();

  Brent_Kung_Scan(shared_data, valid);

  int nfaces = min(*num_faces, MAX_FACES);
  
  #pragma unroll
  for (int i = 0; i < ITEMS_PER_THREAD; i++) {
    int y_local = threadIdx.x * ITEMS_PER_THREAD + i;
    int y_global = y0 + y_local;

    if (y_local < valid) {
      int top = max(0, y_global - R);
      int bottom = min(height - 1, y_global + R);
      int T = max(0, top - y0);
      int B = min(valid - 1, bottom - y0);

      int sum = 0;
      if (T <= B) {
        sum = shared_data[B] - (T > 0 ? shared_data[T - 1] : 0);
      }

      for (int r = top; r < y0; r++) {
        sum += hsum[r * pitch + col];
      }
      for (int r = y0 + valid; r <= bottom; r++) {
        if (r < height) {
          sum += hsum[r * pitch + col];
        }
      }
      
      // Use box_count function to calculate the number of pixels in the blur kernel
      int count = box_count(col, y_global, width, height, R);
      int avg = (count > 0) ? (sum / count) : int(d_in[y_global * pitch + col]);

      bool should_blur = false;
      if (nfaces == 1) {
        int dx = col - blur_x[0];
        int dy = y_global - blur_y[0];
        should_blur = (dx * dx + dy * dy <= distance[0] * distance[0]);
      } else {
        #pragma unroll
        for (int face_idx = 0; face_idx < nfaces; face_idx++) {
          int dx = col - blur_x[face_idx];
          int dy = y_global - blur_y[face_idx];
          if (dx * dx + dy * dy <= distance[face_idx] * distance[face_idx]) {
            should_blur = true;
            break;
          }
        }
      }
      d_out[y_global * pitch + col] = should_blur ? static_cast<uchar>(avg) : d_in[y_global * pitch + col];
    }
  }
}

//==========================================================================//

// optimized kernel with advanced block-level reductions
void Blur_Brent_Kung(cv::Mat& frame, int width, int height, int frames, int num_pixels, uchar* hr_in, uchar* hg_in, uchar* hb_in, 
              uchar* hr_out, uchar* hg_out, uchar* hb_out, uchar* dr_in, uchar* dg_in, uchar* db_in, 
              uchar* dr_out, uchar* dg_out, uchar* db_out) {

  hipStream_t streams[3];
  for (int i = 0; i < 3; i++) {
    hipStreamCreate(&streams[i]);
  }

  const int W = width;
  const int H = height;
  const int P = width;
  size_t bytesI = W * H * sizeof(int);

  int *d_hsum_r, *d_hsum_g, *d_hsum_b;
  hipMalloc(&d_hsum_r, bytesI);
  hipMalloc(&d_hsum_g, bytesI);
  hipMalloc(&d_hsum_b, bytesI);

  dim3 bh(BLOCK_THREADS), gv(W, (H + BLOCK_THREADS * ITEMS_PER_THREAD - 1) / (BLOCK_THREADS * ITEMS_PER_THREAD));
  dim3 gh((W + (BLOCK_THREADS * ITEMS_PER_THREAD - 1)) / (BLOCK_THREADS * ITEMS_PER_THREAD), H);

  // Asynchronous operations for RGB channels using CUB kernel
  hipMemcpyAsync(dr_in, hr_in, num_pixels * sizeof(uchar), hipMemcpyHostToDevice, streams[0]);
  hipMemcpyAsync(dg_in, hg_in, num_pixels * sizeof(uchar), hipMemcpyHostToDevice, streams[1]);
  hipMemcpyAsync(db_in, hb_in, num_pixels * sizeof(uchar), hipMemcpyHostToDevice, streams[2]);

  // Horizontal pass - compute row sums only
  BoxBlurHorizontal<BLUR_SIZE><<<gh, bh, 0, streams[0]>>>(dr_in, W, H, P, d_hsum_r);
  BoxBlurHorizontal<BLUR_SIZE><<<gh, bh, 0, streams[1]>>>(dg_in, W, H, P, d_hsum_g);
  BoxBlurHorizontal<BLUR_SIZE><<<gh, bh, 0, streams[2]>>>(db_in, W, H, P, d_hsum_b);

  // Vertical pass - compute final blur using box_count function for pixel counts
  BoxBlurVertical<BLUR_SIZE><<<gv, bh, 0, streams[0]>>>(d_hsum_r, dr_in, dr_out, W, H, P);
  BoxBlurVertical<BLUR_SIZE><<<gv, bh, 0, streams[1]>>>(d_hsum_g, dg_in, dg_out, W, H, P);
  BoxBlurVertical<BLUR_SIZE><<<gv, bh, 0, streams[2]>>>(d_hsum_b, db_in, db_out, W, H, P);

  hipMemcpyAsync(hr_out, dr_out, num_pixels * sizeof(uchar), hipMemcpyDeviceToHost, streams[0]);
  hipMemcpyAsync(hg_out, dg_out, num_pixels * sizeof(uchar), hipMemcpyDeviceToHost, streams[1]);
  hipMemcpyAsync(hb_out, db_out, num_pixels * sizeof(uchar), hipMemcpyDeviceToHost, streams[2]);

  // Synchronize all streams
  for (int i = 0; i < 3; i++) {
    hipStreamSynchronize(streams[i]);
    hipStreamDestroy(streams[i]);
  }

  // Free temporary buffers (only sum arrays now)
  hipFree(d_hsum_r);
  hipFree(d_hsum_g);
  hipFree(d_hsum_b);

  // Update frame data
  #pragma omp parallel for collapse(2)
  for (int i = 0; i < height; i++) {
    for (int j = 0; j < width; j++) {
      cv::Vec3b& pixel = frame.at<cv::Vec3b>(i, j);
      pixel[2] = hr_out[i * width + j];
      pixel[1] = hg_out[i * width + j];
      pixel[0] = hb_out[i * width + j];
    }
  }
}

// Explicit template instantiation to avoid linking issues
template __global__ void BoxBlurHorizontal<BLUR_SIZE, 128, 8>(const uchar*, int, int, int, int*);
template __global__ void BoxBlurVertical<BLUR_SIZE, 128, 8>(const int*, const uchar*, uchar*, int, int, int);
